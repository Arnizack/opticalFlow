// errorChecking.cu
#include "errorChecking.cuh"
#include <stdio.h>

hipError_t checkAndPrint(const char* name, int sync) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        const char* errorMessage = hipGetErrorString(err);
        fprintf(stderr, "CUDA error check \"%s\" returned ERROR code: %d (%s) %s \n", name, err, errorMessage, (sync) ? "after sync" : "");
    }
    else if (PRINT_ON_SUCCESS) {
        printf("CUDA error check \"%s\" executed successfully %s\n", name, (sync) ? "after sync" : "");
    }
    return err;
}

hipError_t checkCUDAError(const char* name, int sync) {
    hipError_t err = hipSuccess;
    if (sync || FORCE_SYNC_GPU) {
        err = checkAndPrint(name, 0);
        hipDeviceSynchronize();
        err = checkAndPrint(name, 1);
    }
    else {
        err = checkAndPrint(name, 0);
    }
    return err;
}